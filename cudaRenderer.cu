#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <vector>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cudaRenderer.h"
#include "image.h"
#include "sceneLoader.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
////////////////////////////////////////////////////////////////////////////////////


typedef struct {

	SceneName sceneName;

	int length;
	int width;

	float time_step_size;
	float diff_const;

	int numParticles;
	int size;      

	float** vel_x;
	float** vel_y;

	float** temp_vel_x;
	float** temp_vel_y;

	float** pre_x;
	float** pre_y;

	float** temp_pre_x;
	float** temp_pre_y;

	float** grad_x;
	float** grad_y;

	float** divergence;
	bool** particle;
    bool** temp_particle;

	int imageWidth;
    int imageHeight;
    float* imageData;

} GlobalConstants;

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

__global__ void kernelAdvanceWaterCube(){


}


////////////////////////////////////////////////////////////////////////////////////////
// Cuda Renderer Class
////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {

	image = NULL;
    box = NULL;
    sceneName = WATER_CUBE;

	// length = 0;
	// width = 0;

	// time_step_size = 0;
	// diff_const = 0;

	// numParticles = 0;
	// size = 0;

	// vel_x = NULL;
	// vel_y = NULL;

	// temp_vel_x = NULL;
	// temp_vel_y = NULL;

	// pre_x = NULL;
	// pre_y = NULL;

	// temp_pre_x = NULL;
	// temp_pre_y = NULL;

	// grad_x = NULL;
	// grad_y = NULL;

	// divergence = NULL;
	// particle = NULL;

	cudaDevice_imageData = NULL;    

	cudaDevice_vel_x = NULL;
	cudaDevice_vel_y = NULL;

	cudaDevice_temp_vel_x = NULL;
	cudaDevice_temp_vel_y = NULL;

	cudaDevice_pre_x = NULL;
	cudaDevice_pre_y = NULL;

	cudaDevice_temp_pre_x = NULL;
	cudaDevice_temp_pre_y = NULL;

	cudaDevice_grad_x = NULL;
	cudaDevice_grad_y = NULL;

	cudaDevice_divergence = NULL;
	cudaDevice_particle = NULL;
    cudaDevice_temp_particle = NULL;

}

CudaRenderer::~CudaRenderer(){

	if(image) {
		delete image;
	}

	delete [] box->vel_x;
	delete [] box->vel_y;
	delete [] box->temp_vel_x;
	delete [] box->temp_vel_y;
	delete [] box->pre_x;
	delete [] box->pre_y;
	delete [] box->temp_pre_x;
	delete [] box->temp_pre_y;
	delete [] box->particle;
    delete [] box->temp_particle;
	delete [] box->divergence;
	delete [] box->grad_x;
	delete [] box->grad_y;
    delete [] box;

	hipFree(cudaDevice_vel_x);
	hipFree(cudaDevice_vel_y);
	hipFree(cudaDevice_temp_vel_x);
	hipFree(cudaDevice_temp_vel_y);
	hipFree(cudaDevice_pre_x);
	hipFree(cudaDevice_pre_y);
	hipFree(cudaDevice_temp_pre_x);
	hipFree(cudaDevice_temp_pre_y);
	hipFree(cudaDevice_particle);
    hipFree(cudaDevice_temp_particle);
	hipFree(cudaDevice_divergence);
	hipFree(cudaDevice_grad_x);
	hipFree(cudaDevice_grad_y);
	hipFree(cudaDevice_imageData);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDevice_imageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
   
    hipDeviceSynchronize();
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    // loadWaterScene(length, width, time_step_size, diff_const, numParticles, 
    //     size, sceneName, vel_x, vel_y, temp_vel_x, temp_vel_y, pre_x, pre_y, 
    //     temp_pre_x, temp_pre_y, particle, divergence, grad_x, grad_y);

    box = loadWaterScene();

}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 780") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 780.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    int length = box->length;
    int width = box->width;

    hipMalloc(&cudaDevice_vel_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_vel_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_vel_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_vel_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_pre_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_pre_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_pre_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_pre_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_grad_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_grad_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_particle, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_particle, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_divergence, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_imageData, sizeof(float) * 4 * length * width);

    hipMemcpy(cudaDevice_vel_x, box->vel_x, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_vel_y, box->vel_y, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_vel_x, box->temp_vel_x, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_vel_y, box->temp_vel_y, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_pre_x, box->pre_x, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_pre_y, box->pre_x, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_pre_x, box->temp_pre_x, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_pre_y, box->temp_pre_y, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_grad_x, box->grad_x, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_grad_y, box->grad_y, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_particle, box->particle, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_particle, box->temp_particle, sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_divergence, box->divergence, sizeof(float) * length * width, hipMemcpyHostToDevice);
    

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;

    params.sceneName = sceneName;

	params.length = length;
	params.width = width;

	params.time_step_size = box->time_step_size;
	params.diff_const = box->diff_const;

	params.numParticles = box->numParticles;
	params.size = box->size;      

	params.vel_x = cudaDevice_vel_x;
	params.vel_y = cudaDevice_vel_y;

	params.temp_vel_x = cudaDevice_temp_vel_x;
	params.temp_vel_y = cudaDevice_temp_vel_y;

	params.pre_x = cudaDevice_pre_x;
	params.pre_y = cudaDevice_pre_y;

	params.temp_pre_x = cudaDevice_temp_pre_x;
	params.temp_pre_y = cudaDevice_temp_pre_y;

	params.grad_x = cudaDevice_grad_x;
	params.grad_y = cudaDevice_grad_y;

	params.divergence = cudaDevice_divergence;
	params.particle = cudaDevice_particle;
    params.temp_particle = cudaDevice_temp_particle;

	params.imageHeight = image->height;
	params.imageWidth = image->width;
	params.imageData = cudaDevice_imageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));
}


void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((42 + blockDim.x - 1) / blockDim.x);

    if(sceneName == WATER_CUBE) {
    	kernelAdvanceWaterCube<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {

    //using a 2d block
    dim3 blockDim(BLOCKDIM,BLOCKDIM);
    //splitting the image into a 2d grid of 2d blocks
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);
    printf("In Host\n");

    //kernelRender<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();

}