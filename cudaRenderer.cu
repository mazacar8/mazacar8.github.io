#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <vector>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <iostream>

#include "cudaRenderer.h"
#include "image.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
////////////////////////////////////////////////////////////////////////////////////

typedef struct {

    SceneName sceneName;

    int length;
    int width;

    float time_step_size;
    float diff_const;

    int numParticles;
    int size;      

    float** vel_x;
    float** vel_y;

    float** temp_vel_x;
    float** temp_vel_y;

    float** pre_x;
    float** pre_y;

    float** temp_pre_x;
    float** temp_pre_y;

    float** grad_x;
    float** grad_y;

    float** divergence;
    bool** particle;
    bool** temp_particle;

    int imageWidth;
    int imageHeight;
    float* imageData;

} GlobalConstants;

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants gpuParams;

__global__ void kernelAdvection(){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    int old_x, old_y;

    if(index_x >= 0 and index_x < gpuParams.length and index_y >= 0 or index_y < gpuParams.width){

        old_x = round(index_x - gpuParams.temp_vel_x[index_y][index_x]*gpuParams.time_step_size);
        old_y = round(index_y - gpuParams.temp_vel_y[index_y][index_x]*gpuParams.time_step_size);

        if(old_x < LENGTH and old_x >= 0 and old_y < WIDTH and old_y >= 0 and gpuParams.particle[index_y][index_x]){

            gpuParams.temp_particle[index_y][index_x] = gpuParams.particle[old_y][old_x];

            if(gpuParams.temp_particle[index_y][index_x]){
                gpuParams.temp_vel_x[index_y][index_x] = gpuParams.vel_x[old_y][old_x];
                gpuParams.vel_x[index_y][index_x] = gpuParams.vel_x[old_y][old_x];
                gpuParams.temp_vel_y[index_y][index_x] = gpuParams.vel_y[old_y][old_x];
                gpuParams.vel_y[index_y][index_x] = gpuParams.vel_y[old_y][old_x];
            }
        }

        __syncthreads();

        gpuParams.particle[index_y][index_x] = gpuParams.temp_particle[index_y][index_x];
    }
}

__global__ void kernelDiffusion(){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if(index_x >= 1 and index_x < gpuParams.length - 1 and index_y >= 1 or index_y < gpuParams.width - 1){
        float alpha = (gpuParams.length) * (gpuParams.width)/(gpuParams.time_step_size * gpuParams.diff_const);
        float beta = alpha + 4;

        float sumx, sumy;

        for(int iter = 0; iter < DIFF_ITER; iter++) {

            gpuParams.temp_vel_x[index_y][index_x] = gpuParams.vel_x[index_y][index_x];
            gpuParams.temp_vel_y[index_y][index_x] = gpuParams.vel_y[index_y][index_x];

            sumx = gpuParams.temp_vel_x[index_y - 1][index_x] + gpuParams.temp_vel_x[index_y+1][index_x] +
                   gpuParams.temp_vel_x[index_y][index_x - 1] + gpuParams.temp_vel_x[index_y][index_x + 1];
            sumy = gpuParams.temp_vel_y[index_y - 1][index_x] + gpuParams.temp_vel_y[index_y+1][index_x] +
                   gpuParams.temp_vel_y[index_y][index_x - 1] + gpuParams.temp_vel_y[index_y][index_x + 1];

            gpuParams.vel_x[index_y][index_x] = (sumx + alpha*gpuParams.temp_vel_x[index_y][index_x])/beta;
            gpuParams.vel_y[index_y][index_x] = (sumy + alpha*gpuParams.temp_vel_y[index_y][index_x])/beta;

            __syncthreads();
        }
    }
}

__global__ void kernelProjection(){
    
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    float alpha = gpuParams.length * gpuParams.width;
    float beta = 4;

    gpuParams.divergence[index_y][index_x] = (gpuParams.temp_vel_y[index_y - 1][index_x] - gpuParams.temp_vel_y[index_y+1][index_x] +
                   gpuParams.temp_vel_x[index_y][index_x - 1] - gpuParams.temp_vel_x[index_y][index_x + 1])/2;

    gpuParams.pre_x[index_y][index_x] = 0.0f;
    gpuParams.pre_y[index_y][index_x] = 0.0f;

    __syncthreads();

    if(index_x >= 1 and index_x < gpuParams.length - 1 and index_y >= 1 or index_y < gpuParams.width - 1){

        float sumx, sumy;

        for(int iter = 0; iter < DIFF_ITER; iter++) {

            gpuParams.temp_pre_x[index_y][index_x] = gpuParams.pre_y[index_y][index_x];
            gpuParams.temp_pre_y[index_y][index_x] = gpuParams.pre_y[index_y][index_x];

            sumx = gpuParams.temp_pre_x[index_y - 1][index_x] + gpuParams.temp_pre_x[index_y+1][index_x] +
                   gpuParams.temp_pre_x[index_y][index_x - 1] + gpuParams.temp_pre_x[index_y][index_x + 1];
            sumy = gpuParams.temp_pre_y[index_y - 1][index_x] + gpuParams.temp_pre_y[index_y+1][index_x] +
                   gpuParams.temp_pre_y[index_y][index_x - 1] + gpuParams.temp_pre_y[index_y][index_x + 1];

            gpuParams.pre_x[index_y][index_x] = (sumx + alpha*gpuParams.divergence[index_y][index_x])/beta;
            gpuParams.pre_y[index_y][index_x] = (sumy + alpha*gpuParams.divergence[index_y][index_x])/beta;

            __syncthreads();
        }
    }
}

__global__ void kernelGradientComputation(){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    gpuParams.grad_x[index_y][index_x] = (gpuParams.pre_x[index_y+1][index_x] - gpuParams.pre_x[index_y-1][index_x])/2;
    gpuParams.grad_y[index_y][index_x] = (gpuParams.pre_y[index_y][index_x+1] - gpuParams.pre_y[index_y][index_x-1])/2;

    gpuParams.temp_vel_x[index_y][index_x] -= gpuParams.grad_x[index_y][index_x];
    gpuParams.temp_vel_y[index_y][index_x] -= gpuParams.grad_y[index_y][index_x]; 

    gpuParams.vel_x[index_y][index_x] -= gpuParams.grad_x[index_y][index_x];
    gpuParams.vel_y[index_y][index_x] -= gpuParams.grad_y[index_y][index_x]; 
    
}

__device__ __inline__ void
shadePixel(float4* imgPtr){

    float r = 1.f;
    float g = 0.f;
    float b = 0.f;
    float a = 1.f;    
    float4 color = make_float4(r, g, b, a);
    *imgPtr = color;

}

__global__ void kernelRender(){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if(index_x < gpuParams.width and index_y < gpuParams.length){

        int imageWidth = gpuParams.imageWidth;
        int offset = 4 * (index_y * imageWidth + index_x);

        float4* imgPtr = (float4 *) (&gpuParams.imageData[offset]);

        if(gpuParams.particle[index_x][index_y]){
            shadePixel(imgPtr);
        }
    }

}

__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = gpuParams.imageWidth;
    int height = gpuParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&gpuParams.imageData[offset]) = value;
}

////////////////////////////////////////////////////////////////////////////////////////
// Cuda Renderer Class
////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {

    image = NULL;
    box = NULL;
    sceneName = WATER_CUBE;

    // length = 0;
    // width = 0;

    // time_step_size = 0;
    // diff_const = 0;

    // numParticles = 0;
    // size = 0;

    // vel_x = NULL;
    // vel_y = NULL;

    // temp_vel_x = NULL;
    // temp_vel_y = NULL;

    // pre_x = NULL;
    // pre_y = NULL;

    // temp_pre_x = NULL;
    // temp_pre_y = NULL;

    // grad_x = NULL;
    // grad_y = NULL;

    // divergence = NULL;
    // particle = NULL;

    cudaDevice_imageData = NULL;    

    cudaDevice_vel_x = NULL;
    cudaDevice_vel_y = NULL;

    cudaDevice_temp_vel_x = NULL;
    cudaDevice_temp_vel_y = NULL;

    cudaDevice_pre_x = NULL;
    cudaDevice_pre_y = NULL;

    cudaDevice_temp_pre_x = NULL;
    cudaDevice_temp_pre_y = NULL;

    cudaDevice_grad_x = NULL;
    cudaDevice_grad_y = NULL;

    cudaDevice_divergence = NULL;
    cudaDevice_particle = NULL;
    cudaDevice_temp_particle = NULL;

}

CudaRenderer::~CudaRenderer(){

    if(image) {
        delete image;
    }

    delete [] box->vel_x;
    delete [] box->vel_y;
    delete [] box->temp_vel_x;
    delete [] box->temp_vel_y;
    delete [] box->pre_x;
    delete [] box->pre_y;
    delete [] box->temp_pre_x;
    delete [] box->temp_pre_y;
    delete [] box->particle;
    delete [] box->temp_particle;
    delete [] box->divergence;
    delete [] box->grad_x;
    delete [] box->grad_y;
    delete [] box;

    hipFree(cudaDevice_vel_x);
    hipFree(cudaDevice_vel_y);
    hipFree(cudaDevice_temp_vel_x);
    hipFree(cudaDevice_temp_vel_y);
    hipFree(cudaDevice_pre_x);
    hipFree(cudaDevice_pre_y);
    hipFree(cudaDevice_temp_pre_x);
    hipFree(cudaDevice_temp_pre_y);
    hipFree(cudaDevice_particle);
    hipFree(cudaDevice_temp_particle);
    hipFree(cudaDevice_divergence);
    hipFree(cudaDevice_grad_x);
    hipFree(cudaDevice_grad_y);
    hipFree(cudaDevice_imageData);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);

}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDevice_imageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
   
    hipDeviceSynchronize();
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    box = FluidBoxCreate2D(LENGTH,WIDTH,DT);

}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 780") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 780.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    int length = box->length;
    int width = box->width;

    hipMalloc(&cudaDevice_imageData, sizeof(float) * 4 * image->width * image->height);
    hipMalloc(&cudaDevice_vel_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_vel_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_vel_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_vel_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_pre_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_pre_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_pre_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_temp_pre_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_grad_x, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_grad_y, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_divergence, sizeof(float) * length * width);
    hipMalloc(&cudaDevice_particle, sizeof(bool) * length * width);
    hipMalloc(&cudaDevice_temp_particle, sizeof(bool) * length * width);

    hipMemcpy(cudaDevice_vel_x, box->vel_x[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_vel_y, box->vel_y[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_vel_x, box->temp_vel_x[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_vel_y, box->temp_vel_y[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_pre_x, box->pre_x[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_pre_y, box->pre_x[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_pre_x, box->temp_pre_x[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_pre_y, box->temp_pre_y[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_grad_x, box->grad_x[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_grad_y, box->grad_y[0], sizeof(float) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_particle, box->particle[0], sizeof(bool) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_temp_particle, box->temp_particle[0], sizeof(bool) * length * width, hipMemcpyHostToDevice);
    hipMemcpy(cudaDevice_divergence, box->divergence[0], sizeof(float) * length * width, hipMemcpyHostToDevice);    

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;

    params.sceneName = sceneName;

    params.length = length;
    params.width = width;

    params.time_step_size = box->time_step_size;
    params.diff_const = box->diff_const;

    params.numParticles = box->numParticles;
    params.size = box->size;      

    params.vel_x = cudaDevice_vel_x;
    params.vel_y = cudaDevice_vel_y;

    params.temp_vel_x = cudaDevice_temp_vel_x;
    params.temp_vel_y = cudaDevice_temp_vel_y;

    params.pre_x = cudaDevice_pre_x;
    params.pre_y = cudaDevice_pre_y;

    params.temp_pre_x = cudaDevice_temp_pre_x;
    params.temp_pre_y = cudaDevice_temp_pre_y;

    params.grad_x = cudaDevice_grad_x;
    params.grad_y = cudaDevice_grad_y;

    params.divergence = cudaDevice_divergence;
    params.particle = cudaDevice_particle;
    params.temp_particle = cudaDevice_temp_particle;

    params.imageHeight = image->height;
    params.imageWidth = image->width;
    params.imageData = cudaDevice_imageData;

    hipMemcpyToSymbol(HIP_SYMBOL(gpuParams), &params, sizeof(GlobalConstants));
}


void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(BLOCKDIM, BLOCKDIM);
    dim3 gridDim((LENGTH + blockDim.x - 1)/blockDim.x, (
                  WIDTH + blockDim.y - 1)/blockDim.y);

    kernelAdvection<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelDiffusion<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelProjection<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelGradientComputation<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}
void
CudaRenderer::render() {

    //using a 2d block
    dim3 blockDim(BLOCKDIM,BLOCKDIM);
    //splitting the image into a 2d grid of 2d blocks
    dim3 gridDim(
        (LENGTH + blockDim.x - 1) / blockDim.x,
        (WIDTH + blockDim.y - 1) / blockDim.y);
    printf("In Host\n");

    kernelRender<<<gridDim, blockDim>>>();

    hipDeviceSynchronize();

}